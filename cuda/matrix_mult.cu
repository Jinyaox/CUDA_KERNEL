
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

void matrix_init(int* vec,int size,int n){
    for(int i=0;i<size;i++){
        vec[i]=n;
    }
    return;
}

__global__ void mat_mult(int* a,int* b,int* out,int n){
    int row=(blockIdx.y * blockDim.y)+threadIdx.y; //y determines the vertical rows 
    int col=(blockIdx.x * blockDim.x)+threadIdx.x; //x determines the horizontal columns
    int temp=0; //does not matter becuase each time there will be one 
    if((row<n)&&(col<n)){
        for(int i=0;i<n;i++){
            temp+=a[row*n+i]*b[i*n+col];
        }
        out[row*n+col]=temp;
    }
}

/*void matmult(int*a,int *b,int n){ //this is how regular 2D array multiplication look like
    for(int i=0;i<n;i++){
        for(int j=0;j<n;j++){
            for(int k=0;k<n;k++){//traversing part
                //a is the row
                out[i*n+j]=a[i*n+k]*b[k*n+j];
            }
        }
    }
}*/


void verify_result(vector<int> &a, vector<int> &b, vector<int> &c, int N) {
  // For every row...
  for (int i = 0; i < N; i++) {
    // For every column...
    for (int j = 0; j < N; j++) {
      // For every element in the row-column pair
      int tmp = 0;
      for (int k = 0; k < N; k++) {
        // Accumulate the partial results
        tmp += a[i * N + k] * b[k * N + j];
      }

      // Check against the CPU result
      assert(tmp == c[i * N + j]);
    }
  }
}


//this is the vector add in unified memory
int main() {
  // Matrix size of 1024 x 1024;
  int N = 1 << 10;

  // Size (in bytes) of matrix
  size_t bytes = N * N * sizeof(int);

  // Host vectors
  vector<int> h_a(N * N);
  vector<int> h_b(N * N);
  vector<int> h_c(N * N);

  // Initialize matrices
  generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
  generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

  // Allocate device memory
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  // Copy data to the device
  hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

  // Threads per CTA dimension
  int THREADS = 32;

  // Blocks per grid dimension (assumes THREADS divides N evenly)
  int BLOCKS = N / THREADS;

  // Use dim3 structs for block  and grid dimensions
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  // Launch kernel
  mat_mult<<<blocks, threads>>>(d_a, d_b, d_c, N);

  // Copy back to the host
  hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

  // Check result
  verify_result(h_a, h_b, h_c, N);

  cout << "COMPLETED SUCCESSFULLY\n";

  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}